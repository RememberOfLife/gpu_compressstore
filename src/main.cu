#include <bitset>
#include <hipcub/hipcub.hpp>
#include <cstdint>
#include <iostream>
#include <stdio.h>

#include "benchmark_data.cuh"
#include "cuda_try.cuh"
#include "kernels/kernel_4pass.cuh"
#include "kernels/kernel_singlethread.cuh"

template <typename T>
void gpu_buffer_print(T* d_buffer, uint32_t offset, uint32_t count) {
    T* h_buffer = static_cast<T*>(malloc(count*sizeof(T)));
    CUDA_TRY(hipMemcpy(h_buffer, d_buffer, count*sizeof(T), hipMemcpyDeviceToHost));
    for (int i = offset; i < offset+count; i++) {
        std::bitset<sizeof(T)*8> bits(h_buffer[i]);
        std::cout << bits << " - " << h_buffer[i] << "\n";
    }
    free(h_buffer);
}

int main()
{
    benchmark_data<uint64_t> bdata(true, 128); // 2<<17 = 1MiB worth of elems (2<<27 = 2GiB)
    bdata.generate_mask(MASKTYPE_UNIFORM, 0.5);

    
    // 4 pass algo
    uint32_t chunk_length = 32;
    uint32_t pass1_blockcount = 0;
    uint32_t pass1_threadcount = 256;
    //uint32_t pass2_blockcount = 0;
    //uint32_t pass2_threadcount = 256;

    uint32_t chunk_count = bdata.count / chunk_length;
    uint16_t* d_pss; // prefix sum scan buffer on device
    CUDA_TRY(hipMalloc(&d_pss, chunk_count*sizeof(uint16_t)));
    iovRow* d_iov; // intermediate optimization vector
    CUDA_TRY(hipMalloc(&d_iov, chunk_count*sizeof(iovRow)));
    uint32_t* d_pss_total; // pss total
    CUDA_TRY(hipMalloc(&d_pss_total, sizeof(uint32_t)));
    // #1: pop count per chunk and populate IOV
    launch_4pass_popc(pass1_blockcount, pass1_threadcount, bdata.d_mask, d_pss, d_iov, chunk_length, chunk_count);
    // #2: prefix sum scan (for partial trees)
    //launch_4pass_pss(pass2_blockcount, pass2_threadcount, d_pss, chunk_count, d_pss_total);
    {
        // use cub pss for now
        uint16_t* d_pss_tmp;
        CUDA_TRY(hipMalloc(&d_pss_tmp, chunk_count*sizeof(uint16_t)));
        void* d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        CUDA_TRY(hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_pss, d_pss_tmp, chunk_count));
        CUDA_TRY(hipMalloc(&d_temp_storage, temp_storage_bytes));
        CUDA_TRY(hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_pss, d_pss_tmp, chunk_count));
        CUDA_TRY(hipFree(d_temp_storage));
        //uint16_t* d_pss_die = d_pss;
        //d_pss = d_pss_tmp;
        CUDA_TRY(hipMemcpy(d_pss, d_pss_tmp, chunk_count*sizeof(uint16_t), hipMemcpyDeviceToDevice));
        CUDA_TRY(hipFree(d_pss_tmp));
    }

    gpu_buffer_print(d_pss, 0, 4);
    gpu_buffer_print(reinterpret_cast<uint32_t*>(d_iov), 0, 4);
    // #3: optimization pass (sort or bucket skip launch)
    // #4: processing of chunks

    // free temporary device resources
    CUDA_TRY(hipFree(d_iov));
    CUDA_TRY(hipFree(d_pss));


    CUDA_TRY(hipMemcpy(bdata.h_output, bdata.d_output, bdata.count*sizeof(uint64_t), hipMemcpyDeviceToHost));
    /*/ print for testing (first 64 elems of input, validation and mask)
    std::bitset<8> maskset(bdata.h_mask[0]);
    std::cout << "maskset: " << maskset << "\n\n";
    for (int i = 0; i < 64; i ++) {
        // mask value for this input
        uint32_t offset32 = i % 8;
        uint32_t base32 = (i-offset32) / 8;
        uint32_t mask32 = reinterpret_cast<uint8_t*>(bdata.h_mask)[base32];
        uint32_t mask = 0b1 & (mask32>>(7-offset32));
        std::cout << mask;
        // print number residing there
        uint64_t num = bdata.h_input[i];
        std::bitset<64> numbs(num);
        std::bitset<64> valid(bdata.h_validation[i]);
        std::bitset<64> gout(bdata.h_output[i]);
        std::cout << " - " << numbs << " - " << valid << " - " << gout << "\n";
    }//*/

    //bdata.validate(bdata.count);

    printf("done");
    return 0;
}
